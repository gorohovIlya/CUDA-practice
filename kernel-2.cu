#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>


__global__ void generateAndFindAKF(size_t* goodSignals, int* sidelobes, int n) {
    size_t signal = blockIdx.x;
    extern __shared__ int s_akf[];

    if (threadIdx.x < n) {
        s_akf[threadIdx.x] = 0;
    }
    __syncthreads();

    int threadBit = ((signal >> threadIdx.x) & 1) ? 1 : -1;
    for (int shift = 0; shift < n; shift++) {
        if (threadIdx.x >= shift) {
            int paired_bit = ((signal >> (threadIdx.x - shift)) & 1) ? 1 : -1;
            atomicAdd(&s_akf[shift], threadBit * paired_bit);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        int max_sidelobe = 0;
        for (int i = 1; i < n; i++) {
            int sidelobe = s_akf[i];
            if (sidelobe < 0) {
                sidelobe *= -1;
            }
            if (sidelobe > max_sidelobe) {
                max_sidelobe = sidelobe;
            }
        }

        if (n > 24) {
            if (max_sidelobe < 3 && (((signal & 0b111) == 0b111) || ((signal >> (n - 3) == 0b111)))) {
                goodSignals[blockIdx.x] = signal;
                sidelobes[blockIdx.x] = max_sidelobe;
        }
            else {
        goodSignals[blockIdx.x] = 0;
        }
    }
        else {
            if (max_sidelobe < 2 && (((signal & 0b111) == 0b111) || ((signal >> (n - 3)) == 0b111))) {
                goodSignals[blockIdx.x] = signal;
                sidelobes[blockIdx.x] = max_sidelobe;
            }
            else {
                goodSignals[blockIdx.x] = 0;
            }
        }
    }
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    const int n = 25;
    const size_t total_signals = 1ull << n;
    FILE* file = fopen("goodSignals.txt", "w");
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    size_t required_mem = total_signals * (sizeof(size_t) + sizeof(int));
    if (required_mem > free_mem) {
        fprintf(stderr, "Not enough GPU memory. Required: %zu MB, Available: %zu MB\n",
            required_mem / (1024 * 1024), free_mem / (1024 * 1024));
        return 1;
    }

    size_t* d_goodSignals = nullptr;
    int* d_sidelobes = nullptr;
    hipMalloc(&d_goodSignals, total_signals * sizeof(size_t));
    hipMalloc(&d_sidelobes, total_signals * sizeof(int));

    hipMemset(d_goodSignals, 0, total_signals * sizeof(size_t));

    printf("Launching %zu blocks with %d threads each...\n", total_signals, n);
    hipEventRecord(start);
    generateAndFindAKF << <total_signals, n, n * sizeof(int) >> > (d_goodSignals, d_sidelobes, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    hipGetLastError();

    size_t* h_goodSignals = new size_t[total_signals];
    int* h_sidelobes = new int[total_signals];
    hipMemcpy(h_goodSignals, d_goodSignals, total_signals * sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_sidelobes, d_sidelobes, total_signals * sizeof(int), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < total_signals; i++) {
        if (h_goodSignals[i]) {
            fprintf(file, "Signal %zu: ", h_goodSignals[i]);
            for (int j = n - 1; j >= 0; j--) {
                fprintf(file, "%d", (h_goodSignals[i] >> j) & 1);
            }
            fprintf(file, " | Max sidelobe: %d\n", h_sidelobes[i]);
        }
    }
    fprintf(file, "\n Execution time: %.2f ms", ms);
    printf("\nResults saved to goodSignals.txt");
    delete[] h_goodSignals;
    delete[] h_sidelobes;
    hipFree(d_goodSignals);
    hipFree(d_sidelobes);
    fclose(file);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
