#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <cmath>
#include <fstream>
#include <string>


__global__ void akf_kernel(size_t* dev_signal, int* dev_max, size_t n)
{

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int* akf = new int[n];
    for (size_t i = 0; i < n; i++) {
        akf[i] = 0;
        for (size_t j = 0; j < n; j++) {
            if (i + j < n) {
                akf[i] += ((dev_signal[idx] >> (i + j) & 1) ? 1 : -1) * ((dev_signal[idx] >> (j) & 1) ? 1 : -1);
            }
        }
    }
    int max1_val = -10000000;
    int max2_val = -10000000;

    for (size_t i = 0; i < n; ++i) {
        if (abs(akf[i]) > max1_val) {
            max2_val = max1_val;
            max1_val = abs(akf[i]);
        }
        else if (abs(akf[i]) > max2_val && abs(akf[i]) != max1_val) {
            max2_val = abs(akf[i]);
        }
    }
    dev_max[idx] = max2_val;
    delete[] akf;
}

std::string intToBinaryString(size_t number, size_t n)
{
    std::string binaryStr;
    while (number > 0) {
        binaryStr.insert(binaryStr.begin(), (number % 2) + '0');
        number /= 2;
    }
    if (binaryStr.empty()) binaryStr = "0";


    while (binaryStr.length() < n) {
        binaryStr.insert(binaryStr.begin(), '0');
    }

    return binaryStr;
}

std::string invertBinaryString(const std::string& binaryStr)
{
    std::string invertedStr;
    for (char ch : binaryStr) {
        invertedStr.push_back(ch == '0' ? '1' : '0');
    }
    return invertedStr;
}

int main()
{
    size_t n = 26;
    size_t N = 1048576 / 8;
    size_t NM = (1 << n) / N;
    int maxD = 10000000;
    size_t bestSignal = 0;
    int* dev_max;
    size_t* dev_signal;
    size_t* signals = new size_t[N];
    int* maxs = new int[N];
    hipMalloc((void**)&dev_signal, N * sizeof(size_t));
    hipMalloc((void**)&dev_max, N * sizeof(int));

    // Добавляем инструменты для замеров времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Записываем старт
    hipEventRecord(start, 0);

    for (size_t k = 0; k < NM; k++)
    {

        for (size_t i = 0; i < N; i++)
        {
            signals[i] = i + k * N;
        }
        hipMemcpy(dev_signal, signals, N * sizeof(size_t), hipMemcpyHostToDevice);

        dim3 threadsPerBlock = dim3(1024);
        dim3 blocksPerGrid = dim3(N / threadsPerBlock.x);
        akf_kernel << <blocksPerGrid, threadsPerBlock >> > (dev_signal, dev_max, n);

        // hipMemcpy(signals, dev_signal, N * sizeof(size_t), hipMemcpyDeviceToHost);
        hipMemcpy(maxs, dev_max, N * sizeof(int), hipMemcpyDeviceToHost);

        for (size_t i = 0; i < N; i++)
        {

            if (maxs[i] < maxD)
            {
                maxD = maxs[i];
                bestSignal = i + k * N;
            }
        }
    }





    hipFree(dev_signal);
    hipFree(dev_max);
    delete[] signals;
    delete[] maxs;


    std::cout << "Best: " << invertBinaryString(intToBinaryString(bestSignal, n)) << std::endl;

    // Фиксируем конец вычисления
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("GPU execution time: %.3f s\n", elapsed_time_ms/1000);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

