#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

__global__ void kernelAKF(unsigned int currSignal, int* akf, int n) {
    int bit_pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (bit_pos < n) {
        int my_bit = ((currSignal >> bit_pos) & 1) ? 1 : -1;
        for (int shift = 0; shift < n; shift++) {
            if (bit_pos >= shift) {
                int paired_bit_pos = bit_pos - shift;
                int paired_bit = ((currSignal >> paired_bit_pos) & 1) ? 1 : -1;
                atomicAdd(&akf[shift], my_bit * paired_bit);
            }
        }
    }
}

__global__ void generateSignals(unsigned int* signals, int n) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total_combinations = 1u << n;

    if (idx < total_combinations) {
        signals[idx] = idx;
    }
}

int main(int argc, char* argv[])
{
    const int N = 32;
    const unsigned int total_signals = 1 << N;
    int* d_akf0;
    unsigned int* d_signals;
    hipMalloc(&d_signals, total_signals * sizeof(unsigned int));
    hipMalloc(&d_akf0, N * sizeof(double));
    int blockSize = 256;
    int gridSize = (total_signals + blockSize - 1) / blockSize;
    generateSignals << <gridSize, blockSize >> > (d_signals, N);
    unsigned int first_signal;
    hipMemcpy(&first_signal, d_signals, sizeof(unsigned int), hipMemcpyDeviceToHost);
    kernelAKF << <1, N >> > (first_signal, d_akf0, N);
    unsigned int* signals = new unsigned int[total_signals];
    int* akf0 = new int[N];
    hipMemcpy(akf0, d_akf0, N * sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(signals, d_signals, total_signals * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(akf0, d_akf0, N * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++) {
        printf("Signal %d: ", i);
        for (int j = N - 1; j >= 0; j--) {

            printf("%d", (signals[i] >> j) & 1);
        }
        printf("\n");
    }
    printf("ACF of 1st signal:\n");
    for (int j = 0; j < N; j++) {
        printf("acf[%d] = %d\n", j, akf0[j]);
    }
    hipFree(d_signals);
    hipFree(d_akf0);
    delete[] signals;
    delete[] akf0;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
